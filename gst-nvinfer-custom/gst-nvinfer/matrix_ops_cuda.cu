// matrix_ops_cuda.cu
// CUDA kernels and wrappers for small matrix ops: multiplication, determinant, transpose, SVD (cuSOLVER)
// Only for 2x2 and 3x3 float matrices (row-major)

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdio.h>

extern "C" {

// Matrix multiplication: C = A * B (row-major, n x n)
__global__ void matmul2x2_kernel(const float* A, const float* B, float* C) {
    int i = threadIdx.x;
    if (i < 4) {
        int row = i / 2, col = i % 2;
        C[i] = A[row * 2 + 0] * B[0 * 2 + col] + A[row * 2 + 1] * B[1 * 2 + col];
    }
}

void matmul2x2_cuda(const float* h_A, const float* h_B, float* h_C) {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, 4 * sizeof(float));
    hipMalloc(&d_B, 4 * sizeof(float));
    hipMalloc(&d_C, 4 * sizeof(float));
    hipMemcpy(d_A, h_A, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, 4 * sizeof(float), hipMemcpyHostToDevice);
    matmul2x2_kernel<<<1, 4>>>(d_A, d_B, d_C);
    hipMemcpy(h_C, d_C, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

// Determinant for 2x2 matrix
__global__ void det2x2_kernel(const float* A, float* det) {
    *det = A[0] * A[3] - A[1] * A[2];
}

void det2x2_cuda(const float* h_A, float* h_det) {
    float *d_A, *d_det;
    hipMalloc(&d_A, 4 * sizeof(float));
    hipMalloc(&d_det, sizeof(float));
    hipMemcpy(d_A, h_A, 4 * sizeof(float), hipMemcpyHostToDevice);
    det2x2_kernel<<<1, 1>>>(d_A, d_det);
    hipMemcpy(h_det, d_det, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_det);
}

// Transpose 2x2
__global__ void transpose2x2_kernel(const float* A, float* At) {
    At[0] = A[0]; At[1] = A[2];
    At[2] = A[1]; At[3] = A[3];
}

void transpose2x2_cuda(const float* h_A, float* h_At) {
    float *d_A, *d_At;
    hipMalloc(&d_A, 4 * sizeof(float));
    hipMalloc(&d_At, 4 * sizeof(float));
    hipMemcpy(d_A, h_A, 4 * sizeof(float), hipMemcpyHostToDevice);
    transpose2x2_kernel<<<1, 1>>>(d_A, d_At);
    hipMemcpy(h_At, d_At, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_At);
}

// TODO: Add 3x3 versions and cuSOLVER SVD wrappers as needed

} // extern "C"
